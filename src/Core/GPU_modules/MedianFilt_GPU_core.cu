#include "hip/hip_runtime.h"
/* This works has been developed at Diamond Light Source Ltd.
*
* Copyright 2020 Daniil Kazantsev
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
* http://www.apache.org/licenses/LICENSE-2.0
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "MedianFilt_GPU_core.h"
#include "shared.h"
#define MAXSTR 100
/* CUDA implementation of the median filtration and dezingering (2D/3D case)
 *
 * Input Parameters:
 * 1. Noisy image/volume
 * 2. kernel_size: The size of the median filter window
 * 3. mu_threshold: if not a zero value then deinzger

 * Output:
 * [1] Filtered or dezingered image/volume
 */
/********************************************************************/
/***************************2D Functions*****************************/
/********************************************************************/

__global__ void kernel(float *Input, float* Output, int offset,  int N, int M, int num_total)
  {
  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  const int j = blockDim.y * blockIdx.y + threadIdx.y;
  const int index = offset + i + N*j;

      if (index < num_total) Output[index] = Input[index] * 100;
  }

__global__ void medfilt1_kernel_2D(float *Input, float* Output, int offset, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
  {
      float ValVec[CONSTVECSIZE_9];
      int i1, j1, i_m, j_m, counter = 0;

      const int i = blockDim.x * blockIdx.x + threadIdx.x;
      const int j = blockDim.y * blockIdx.y + threadIdx.y;
      const int index = offset + i + N*j;

      if (index < num_total && i < N && j < M)	{
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 =  i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
              ValVec[counter++] = Input[offset + i1 + N*j1];
      }}
      //sort_quick(ValVec, 0, CONSTVECSIZE_9); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_9); /* perform sorting */

      if (mu_threshold == 0.0f) {
            Output[index] = ValVec[midval]; /* perform median filtration */
        }
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
      }
      }
  }

__global__ void medfilt2_kernel_2D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        float ValVec[CONSTVECSIZE_25];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_25); /* perform sorting */
        sort_bubble(ValVec, CONSTVECSIZE_25); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt3_kernel_2D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        float ValVec[CONSTVECSIZE_49];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_49); /* perform sorting */
        sort_bubble(ValVec, CONSTVECSIZE_49); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt4_kernel_2D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        float ValVec[CONSTVECSIZE_81];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_81); /* perform sorting */
        sort_bubble(ValVec, CONSTVECSIZE_81); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt5_kernel_2D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        float ValVec[CONSTVECSIZE_121];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_121); /* perform sorting */
        sort_bubble(ValVec, CONSTVECSIZE_121); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt1_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
  {
      unsigned short ValVec[CONSTVECSIZE_9];
      int i1, j1, i_m, j_m, counter = 0;

      const int i = blockDim.x * blockIdx.x + threadIdx.x;
      const int j = blockDim.y * blockIdx.y + threadIdx.y;
      const int index = i + N*j;

      if (index < num_total)	{
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
              ValVec[counter++] = Input[i1 + N*j1];
      }}
      //sort_quick(ValVec, 0, CONSTVECSIZE_9); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_9); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
      }
      }
  }

__global__ void medfilt2_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        unsigned short ValVec[CONSTVECSIZE_25];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_25); /* perform sorting */
        sort_bubble_uint16(ValVec, CONSTVECSIZE_25); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt3_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        unsigned short ValVec[CONSTVECSIZE_49];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_49); /* perform sorting */
        sort_bubble_uint16(ValVec, CONSTVECSIZE_49); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt4_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        unsigned short ValVec[CONSTVECSIZE_81];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_81); /* perform sorting */
        sort_bubble_uint16(ValVec, CONSTVECSIZE_81); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt5_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        unsigned short ValVec[CONSTVECSIZE_121];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_121); /* perform sorting */
        sort_bubble_uint16(ValVec, CONSTVECSIZE_121); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
          }
        }
    }

/********************************************************************/
/***************************3D Functions*****************************/
/********************************************************************/
__global__ void medfilt1_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
  {
      float ValVec[CONSTVECSIZE_27];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_27); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_27); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
  }

__global__ void medfilt2_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_125];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_125); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_125); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt3_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_343];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_343); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_343); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt4_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_729];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_729); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_729); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt5_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_1331];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_1331); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_1331); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt1_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
  {
      float ValVec[CONSTVECSIZE_27];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_27); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_27); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];  }
      }
      return;
  }

__global__ void medfilt2_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_125];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_125); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_125); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt3_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_343];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_343); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_343); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt4_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_729];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_729); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_729); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt5_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_1331];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_1331); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_1331); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }
/* ======================================================================= */

__global__ void medfilt1_pad_kernel_uint16_3D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
  {
      unsigned short ValVec[CONSTVECSIZE_27];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_27); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_27); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];}
      }
      return;
  }

__global__ void medfilt2_pad_kernel_uint16_3D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_125];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_125); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_125); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt3_pad_kernel_uint16_3D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_343];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_343); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_343); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt4_pad_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_729];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_729); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_729); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt5_pad_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_1331];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_1331); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_1331); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt1_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
  {
      unsigned short ValVec[CONSTVECSIZE_27];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_27); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_27); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt2_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_125];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_125); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_125); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt3_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_343];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_343); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_343); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt4_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_729];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_729); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_729); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt5_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_1331];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_1331); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_1331); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }
/****************************************************************************/


/////////////////////////////////////////////////
/////////////// HOST FUNCTION ///////////////////
/////////////////////////////////////////////////
extern "C" int MedianFilt_GPU_main_float32(float *Input, float *Output, int kernel_size, float mu_threshold, int gpu_device, int N, int M, int Z)
{
  int deviceCount = -1; // number of devices
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "No CUDA devices found\n");
       return -1;
   }

        int ImSize, sizefilter_total, kernel_half_size, midval;
        ImSize = N*M*Z;
        float *d_input0, *d_output0;

        /*set GPU device*/
        checkCudaErrors(hipSetDevice(gpu_device));

        const int nStreams = 4;
        const int n = ImSize;
        //const int n = ImSize * nStreams;
        const int streamSize = n / nStreams;
        const int streamBytes = streamSize * sizeof(float);
        const int bytes = n * sizeof(float);

        // create events and streams
        hipStream_t stream[nStreams];
        for (int i = 0; i < nStreams; ++i)
          checkCudaErrors( hipStreamCreate(&stream[i]) );

        // allocate memory on the device
        checkCudaErrors(hipMalloc((void**)&d_input0, bytes));
        checkCudaErrors(hipMalloc((void**)&d_output0, bytes));
        /*
        checkCudaErrors(hipMemcpy(d_input0,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_output0,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));
        */
	if (Z == 1) {
        /*2D case */
        //dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
        //dim3 dimGrid(idivup((N/nStreams),BLKXSIZE2D), idivup((M/nStreams),BLKYSIZE2D));

        const int blockSize = 16;
        dim3 dimBlock(blockSize,blockSize);
        dim3 dimGrid(idivup(streamSize,blockSize), idivup(streamSize,blockSize));

        sizefilter_total = (int)(pow(kernel_size,2));
        kernel_half_size = (int)((kernel_size-1)/2);
        midval = (int)(sizefilter_total/2);

        for (int i = 0; i < nStreams; ++i) {
          int offset = i * streamSize; // calculate an offset for each stream
          /* copy streamed data from host to the device */
          checkCudaErrors( hipMemcpyAsync(&d_input0[offset], &Input[offset],
                                     streamBytes, hipMemcpyHostToDevice,
                                     stream[i]) );
          checkCudaErrors( hipMemcpyAsync(&d_output0[offset], &Input[offset],
                                     streamBytes, hipMemcpyHostToDevice,
                                     stream[i]) );
          // running the kernel
          //kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
          //medfilt1_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
          //medfilt1_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
          //medfilt1_kernel_2D<<<dimGrid, dimBlock, 0, stream[i] >>>(d_input0, d_output0, offset, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
          //medfilt1_kernel_2D<<<numOfBlocks, numOfThreadsPerBlocks, 0, stream[i] >>>(d_input0+offset, d_output0+offset, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
          //simple_kernel<<<dimGrid, dimBlock, 0, stream[i] >>>(d_input0+offset, d_output0+offset, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
          //kernel<<<dimGrid, dimBlock, 0, stream[i]>>>(d_input0, d_output0, offset, N, M, ImSize);
          medfilt1_kernel_2D<<<dimGrid, dimBlock, 0, stream[i] >>>(d_input0, d_output0, offset, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);

          /* copy processed data from device to the host */
          checkCudaErrors( hipMemcpyAsync(&Output[offset], &d_output0[offset],
                                     streamBytes, hipMemcpyDeviceToHost,
                                     stream[i]) );
          checkCudaErrors( hipDeviceSynchronize() );
          }

        /*destroy streams*/
        for (int i = 0; i < nStreams; ++i)
          checkCudaErrors( hipStreamDestroy(stream[i]) );

        /*free GPU memory*/
        checkCudaErrors(hipFree(d_input0));
        checkCudaErrors(hipFree(d_output0));
        /*checkCudaErrors( hipEventElapsedTime(&ms, startEvent, stopEvent) );*/

        /*
        if (kernel_size == 3) medfilt1_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 5) medfilt2_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 7) medfilt3_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 9) medfilt4_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else medfilt5_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        */
        //checkCudaErrors( hipDeviceSynchronize() );
        //checkCudaErrors(hipPeekAtLastError() );
       }
	else {
		    /*3D case*/
        dim3 dimBlock(BLKXSIZE,BLKYSIZE,BLKZSIZE);
        dim3 dimGrid(idivup(N,BLKXSIZE), idivup(M,BLKYSIZE),idivup(Z,BLKXSIZE));
        sizefilter_total = (int)(pow(kernel_size, 3));
        kernel_half_size = (int)((kernel_size-1)/2);
        midval = (int)(sizefilter_total/2);


        if (Z == kernel_size) {
        /* performs operation only on the central frame using all 3D information */
        /*
        if (kernel_size == 3) medfilt1_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 5) medfilt2_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 7) medfilt3_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 9) medfilt4_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else medfilt5_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        */
        }
        else {
        /* Full data (traditional) 3D case */
        /*
        if (kernel_size == 3) medfilt1_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 5) medfilt2_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 7) medfilt3_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 9) medfilt4_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else medfilt5_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        */
        }
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors(hipPeekAtLastError() );
    		}

        /*CHECK(hipMemcpy(Output,d_output0,ImSize*sizeof(float),hipMemcpyDeviceToHost));*/
        //hipDeviceReset();
        return 0;
}

extern "C" int MedianFilt_GPU_main_uint16(unsigned short *Input, unsigned short *Output, int kernel_size, float mu_threshold, int N, int M, int Z)
{
  int deviceCount = -1; // number of devices
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "No CUDA devices found\n");
       return -1;
   }
        int ImSize, sizefilter_total, kernel_half_size, midval;
        unsigned short *d_input, *d_output;
        ImSize = N*M*Z;

        checkCudaErrors(hipMalloc((void**)&d_input,ImSize*sizeof(unsigned short)));
        checkCudaErrors(hipMalloc((void**)&d_output,ImSize*sizeof(unsigned short)));

        checkCudaErrors(hipMemcpy(d_input,Input,ImSize*sizeof(unsigned short),hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_output,Input,ImSize*sizeof(unsigned short),hipMemcpyHostToDevice));

	if (Z == 1) {
        /*2D case */
        dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
        dim3 dimGrid(idivup(N,BLKXSIZE2D), idivup(M,BLKYSIZE2D));
        sizefilter_total = (int)(pow(kernel_size,2));
        kernel_half_size = (int)((kernel_size-1)/2);
        midval = (int)(sizefilter_total/2);

        if (kernel_size == 3) medfilt1_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 5) medfilt2_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 7) medfilt3_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 9) medfilt4_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else medfilt5_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors(hipPeekAtLastError() );
       }
	else {
		    /*3D case*/
        dim3 dimBlock(BLKXSIZE,BLKYSIZE,BLKZSIZE);
        dim3 dimGrid(idivup(N,BLKXSIZE), idivup(M,BLKYSIZE),idivup(Z,BLKXSIZE));
        sizefilter_total = (int)(pow(kernel_size, 3));
        kernel_half_size = (int)((kernel_size-1)/2);
        midval = (int)(sizefilter_total/2);

        if (Z == kernel_size) {
        /* performs operation only on the central frame using all 3D information */
        if (kernel_size == 3) medfilt1_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 5) medfilt2_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 7) medfilt3_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 9) medfilt4_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else medfilt5_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
          }
        else {
        /* Full data (traditional) 3D case */
        if (kernel_size == 3) medfilt1_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 5) medfilt2_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 7) medfilt3_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 9) medfilt4_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else medfilt5_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
            }
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors(hipPeekAtLastError() );
    		}
        checkCudaErrors(hipMemcpy(Output,d_output,ImSize*sizeof(unsigned short),hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(d_input));
        checkCudaErrors(hipFree(d_output));
        return 0;
}
