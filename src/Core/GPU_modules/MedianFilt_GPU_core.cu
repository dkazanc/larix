#include "hip/hip_runtime.h"
/* This works has been developed at Diamond Light Source Ltd.
*
* Copyright 2020 Daniil Kazantsev
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
* http://www.apache.org/licenses/LICENSE-2.0
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "MedianFilt_GPU_core.h"
#include "shared.h"
#define MAXSTR 100
/* CUDA implementation of the median filtration and dezingering (2D/3D case)
 *
 * Input Parameters:
 * 1. Noisy image/volume
 * 2. kernel_size: The size of the median filter window
 * 3. mu_threshold: if not a zero value then deinzger

 * Output:
 * [1] Filtered or dezingered image/volume
 */
/********************************************************************/
/***************************2D Functions*****************************/
/********************************************************************/
__global__ void medfilt1_kernel_2D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
  {
      float ValVec[CONSTVECSIZE_9];
      int i1, j1, i_m, j_m, counter = 0;

      const int i = blockDim.x * blockIdx.x + threadIdx.x;
      const int j = blockDim.y * blockIdx.y + threadIdx.y;
      const int index = i + N*j;

      if (index < num_total)	{
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
              ValVec[counter++] = Input[i1 + N*j1];
      }}
      //sort_quick(ValVec, 0, CONSTVECSIZE_9); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_9); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
      }
      }
  }

__global__ void medfilt2_kernel_2D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        float ValVec[CONSTVECSIZE_25];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_25); /* perform sorting */
        sort_bubble(ValVec, CONSTVECSIZE_25); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt3_kernel_2D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        float ValVec[CONSTVECSIZE_49];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_49); /* perform sorting */
        sort_bubble(ValVec, CONSTVECSIZE_49); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt4_kernel_2D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        float ValVec[CONSTVECSIZE_81];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_81); /* perform sorting */
        sort_bubble(ValVec, CONSTVECSIZE_81); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt5_kernel_2D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        float ValVec[CONSTVECSIZE_121];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_121); /* perform sorting */
        sort_bubble(ValVec, CONSTVECSIZE_121); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt1_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
  {
      unsigned short ValVec[CONSTVECSIZE_9];
      int i1, j1, i_m, j_m, counter = 0;

      const int i = blockDim.x * blockIdx.x + threadIdx.x;
      const int j = blockDim.y * blockIdx.y + threadIdx.y;
      const int index = i + N*j;

      if (index < num_total)	{
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
              ValVec[counter++] = Input[i1 + N*j1];
      }}
      //sort_quick(ValVec, 0, CONSTVECSIZE_9); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_9); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
      }
      }
  }

__global__ void medfilt2_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        unsigned short ValVec[CONSTVECSIZE_25];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_25); /* perform sorting */
        sort_bubble_uint16(ValVec, CONSTVECSIZE_25); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt3_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        unsigned short ValVec[CONSTVECSIZE_49];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_49); /* perform sorting */
        sort_bubble_uint16(ValVec, CONSTVECSIZE_49); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt4_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        unsigned short ValVec[CONSTVECSIZE_81];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_81); /* perform sorting */
        sort_bubble_uint16(ValVec, CONSTVECSIZE_81); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
          }
        }
    }

__global__ void medfilt5_kernel_uint16_2D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int num_total)
    {
        unsigned short ValVec[CONSTVECSIZE_121];
        int i1, j1, i_m, j_m, counter = 0;

        const int i = blockDim.x * blockIdx.x + threadIdx.x;
        const int j = blockDim.y * blockIdx.y + threadIdx.y;
        const int index = i + N*j;

        if (index < num_total)	{
        for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
              i1 = i + i_m;
              if ((i1 < 0) || (i1 >= N)) i1 = i;
              for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
                j1 = j + j_m;
                if ((j1 < 0) || (j1 >= M)) j1 = j;
                ValVec[counter++] = Input[i1 + N*j1];
        }}
        //sort_quick(ValVec, 0, CONSTVECSIZE_121); /* perform sorting */
        sort_bubble_uint16(ValVec, CONSTVECSIZE_121); /* perform sorting */
        if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
        else {
        /* perform dezingering */
        if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
          }
        }
    }

/********************************************************************/
/***************************3D Functions*****************************/
/********************************************************************/
__global__ void medfilt1_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
  {
      float ValVec[CONSTVECSIZE_27];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_27); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_27); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
  }

__global__ void medfilt2_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_125];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_125); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_125); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt3_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_343];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_343); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_343); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt4_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_729];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_729); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_729); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt5_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_1331];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_1331); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_1331); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt1_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
  {
      float ValVec[CONSTVECSIZE_27];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_27); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_27); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];  }
      }
      return;
  }

__global__ void medfilt2_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_125];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_125); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_125); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt3_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_343];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_343); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_343); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt4_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_729];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_729); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_729); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt5_pad_kernel_3D(float *Input, float* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      float ValVec[CONSTVECSIZE_1331];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_1331); /* perform sorting */
      sort_bubble(ValVec, CONSTVECSIZE_1331); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs(Input[index] - ValVec[midval]) >= mu_threshold) Output[index] = ValVec[midval];
        }
      }
      return;
    }
/* ======================================================================= */

__global__ void medfilt1_pad_kernel_uint16_3D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
  {
      unsigned short ValVec[CONSTVECSIZE_27];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_27); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_27); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];}
      }
      return;
  }

__global__ void medfilt2_pad_kernel_uint16_3D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_125];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_125); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_125); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt3_pad_kernel_uint16_3D(unsigned short *Input, unsigned short* Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_343];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_343); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_343); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt4_pad_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_729];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_729); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_729); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt5_pad_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_1331];
      long i1, j1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long index = N*M*kernel_half_size + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  ValVec[counter] = Input[N*M*(kernel_half_size + k_m) + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_1331); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_1331); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt1_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
  {
      unsigned short ValVec[CONSTVECSIZE_27];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_27); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_27); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt2_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_125];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_125); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_125); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt3_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_343];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_343); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_343); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt4_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_729];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_729); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_729); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }

__global__ void medfilt5_kernel_uint16_3D(unsigned short *Input, unsigned short *Output, int kernel_half_size, int sizefilter_total, float mu_threshold, int midval, int N, int M, int Z, int num_total)
    {
      unsigned short ValVec[CONSTVECSIZE_1331];
      long i1, j1, k1, i_m, j_m, k_m, counter;

      const long i = blockDim.x * blockIdx.x + threadIdx.x;
      const long j = blockDim.y * blockIdx.y + threadIdx.y;
      const long k = blockDim.z * blockIdx.z + threadIdx.z;
      const long index = N*M*k + i + N*j;

      if (index < num_total)	{
      counter = 0l;
      for(i_m=-kernel_half_size; i_m<=kernel_half_size; i_m++) {
            i1 = i + i_m;
            if ((i1 < 0) || (i1 >= N)) i1 = i;
            for(j_m=-kernel_half_size; j_m<=kernel_half_size; j_m++) {
              j1 = j + j_m;
              if ((j1 < 0) || (j1 >= M)) j1 = j;
                for(k_m=-kernel_half_size; k_m<=kernel_half_size; k_m++) {
                  k1 = k + k_m;
                  if ((k1 < 0) || (k1 >= Z)) k1 = k;
                  ValVec[counter] = Input[N*M*k1 + i1 + N*j1];
                  counter++;
      }}}
      //sort_quick(ValVec, 0, CONSTVECSIZE_1331); /* perform sorting */
      sort_bubble_uint16(ValVec, CONSTVECSIZE_1331); /* perform sorting */

      if (mu_threshold == 0.0f) Output[index] = ValVec[midval]; /* perform median filtration */
      else {
      /* perform dezingering */
      if (abs((int)(Input[index]) - (int)(ValVec[midval])) >= (int)(mu_threshold)) Output[index] = ValVec[midval];
        }
      }
      return;
    }
/****************************************************************************/


/////////////////////////////////////////////////
/////////////// HOST FUNCTION ///////////////////
/////////////////////////////////////////////////
extern "C" int MedianFilt_GPU_main_float32(float *Input, float *Output, int kernel_size, float mu_threshold, int gpu_device, int N, int M, int Z)
{
  int deviceCount = -1; // number of devices
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "No CUDA devices found\n");
       return -1;
   }

        int ImSize, sizefilter_total, kernel_half_size, midval;
        ImSize = N*M*Z;
        float *d_input0, *d_output0;

        CHECK(hipSetDevice(gpu_device));

        const int nStreams = 4;
        const int n = ImSize;
        //const int n = ImSize * nStreams;
        const int streamSize = n / nStreams;
        const int streamBytes = streamSize * sizeof(float);
        const int bytes = n * sizeof(float);

        // create events and streams
        hipEvent_t startEvent, stopEvent, dummyEvent;
        hipStream_t stream[nStreams];
        CHECK( hipEventCreate(&startEvent) );
        CHECK( hipEventCreate(&stopEvent) );
        CHECK( hipEventCreate(&dummyEvent) );
        for (int i = 0; i < nStreams; ++i)
          CHECK( hipStreamCreate(&stream[i]) );



        //const int NUM_STREAMS = 4;
        //int NUM_STREAMS = 4;
        //hipStream_t streams[NUM_STREAMS];
        //for (int i = 0; i < NUM_STREAMS; i++) { hipStreamCreate(&streams[i]); }

        //hipStream_t stream_one;
        //hipStream_t stream_two;
        //hipStream_t stream_three;
        //hipStream_t stream_four;

        //hipStreamCreate(&stream_one);
        //hipStreamCreate(&stream_two);
        //hipStreamCreate(&stream_three);
        //hipStreamCreate(&stream_four);


        //size_t numBytes  = 4 * 1024 * ImSize;
        //size_t totalMemSize = ImSize * sizeof(float);
        //size_t streamMemSize = totalMemSize/4;

        //CHECK(hipMalloc((void**)&d_input0, totalMemSize/4));
        //CHECK(hipMalloc((void**)&d_input1, totalMemSize/4));
        //CHECK(hipMalloc((void**)&d_input2, totalMemSize/4));
        //CHECK(hipMalloc((void**)&d_input3, totalMemSize/4));

        //CHECK(hipMalloc((void**)&d_output0, totalMemSize/4));
        //CHECK(hipMalloc((void**)&d_output1, totalMemSize/4));
        //CHECK(hipMalloc((void**)&d_output2, totalMemSize/4));
        //CHECK(hipMalloc((void**)&d_output3, totalMemSize/4));

        CHECK(hipMalloc((void**)&d_input0, bytes));
        CHECK(hipMalloc((void**)&d_output0, bytes));
        /*
        */
        /*CHECK(hipMemcpy(d_input0,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));*/
        /*CHECK(hipMemcpy(d_output0,Input,ImSize*sizeof(float),hipMemcpyHostToDevice));*/


	if (Z == 1) {
        /*2D case */
        /*
        dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
        dim3 dimGrid(idivup(N,BLKXSIZE2D), idivup(M,BLKYSIZE2D));
        */
        sizefilter_total = (int)(pow(kernel_size,2));
        kernel_half_size = (int)((kernel_size-1)/2);
        midval = (int)(sizefilter_total/2);

        /*for (int i = 0; i < 4; ++i) {
          int offset = i * streamMemSize;
        */
       //Copy the source image to the device i.e. GPU
        //hipMemcpyAsync(d_input0, Input, (totalMemSize)/4, hipMemcpyHostToDevice, stream_one);
        //hipMemcpyAsync(d_input1, Input + streamMemSize, (totalMemSize)/4, hipMemcpyHostToDevice, stream_two);
        //hipMemcpyAsync(d_input2, Input + (2 * streamMemSize), (totalMemSize)/4, hipMemcpyHostToDevice, stream_three);
        //hipMemcpyAsync(d_input3, Input + (3 * streamMemSize), (totalMemSize)/4, hipMemcpyHostToDevice, stream_four);

        //RESULT copy: GPU to CPU
        /*
        hipMemcpyAsync(Output, d_input0, totalMemSize/4, hipMemcpyDeviceToHost, stream_one);
        hipMemcpyAsync(Output + streamMemSize, d_input1, totalMemSize/4, hipMemcpyDeviceToHost, stream_three);
        hipMemcpyAsync(Output + (2 * streamMemSize), d_input2, totalMemSize/4, hipMemcpyDeviceToHost, stream_three);
        hipMemcpyAsync(Output + (3 * streamMemSize), d_input3, totalMemSize/4, hipMemcpyDeviceToHost, stream_four);
        */

        // wait for results
        //hipStreamSynchronize(stream_one);
        //hipStreamSynchronize(stream_two);
        //hipStreamSynchronize(stream_three);
        //hipStreamSynchronize(stream_four);

        CHECK( hipEventRecord(startEvent,0) );
        for (int i = 0; i < nStreams; ++i) {
          int offset = i * streamSize;
          CHECK( hipMemcpyAsync(&d_input0[offset], &Input[offset],
                                     streamBytes, hipMemcpyHostToDevice,
                                     stream[i]) );
          CHECK( hipMemcpyAsync(&d_output0[offset], &Input[offset],
                                     streamBytes, hipMemcpyHostToDevice,
                                     stream[i]) );
          //kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
          //medfilt1_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
          //medfilt1_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
          CHECK( hipMemcpyAsync(&Output[offset], &d_output0[offset],
                                     streamBytes, hipMemcpyDeviceToHost,
                                     stream[i]) );
        }
        CHECK( hipEventRecord(stopEvent, 0) );
        CHECK( hipEventSynchronize(stopEvent) );

        /*CHECK( hipEventElapsedTime(&ms, startEvent, stopEvent) );*/

        /*
        if (kernel_size == 3) medfilt1_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 5) medfilt2_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 7) medfilt3_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 9) medfilt4_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else medfilt5_kernel_2D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        */
        //checkCudaErrors( hipDeviceSynchronize() );
        //checkCudaErrors(hipPeekAtLastError() );
       }
	else {
		    /*3D case*/
        dim3 dimBlock(BLKXSIZE,BLKYSIZE,BLKZSIZE);
        dim3 dimGrid(idivup(N,BLKXSIZE), idivup(M,BLKYSIZE),idivup(Z,BLKXSIZE));
        sizefilter_total = (int)(pow(kernel_size, 3));
        kernel_half_size = (int)((kernel_size-1)/2);
        midval = (int)(sizefilter_total/2);


        if (Z == kernel_size) {
        /* performs operation only on the central frame using all 3D information */
        /*
        if (kernel_size == 3) medfilt1_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 5) medfilt2_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 7) medfilt3_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 9) medfilt4_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else medfilt5_pad_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        */
        }
        else {
        /* Full data (traditional) 3D case */
        /*
        if (kernel_size == 3) medfilt1_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 5) medfilt2_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 7) medfilt3_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 9) medfilt4_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else medfilt5_kernel_3D<<<dimGrid,dimBlock>>>(d_input0, d_output0, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        */
        }
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors(hipPeekAtLastError() );
    		}

        /*CHECK(hipMemcpy(Output,d_output0,ImSize*sizeof(float),hipMemcpyDeviceToHost));*/
        /*
        CHECK(hipFree(d_input0));
        CHECK(hipFree(d_input1));
        CHECK(hipFree(d_input2));
        CHECK(hipFree(d_input3));

        CHECK(hipFree(d_output0));
        CHECK(hipFree(d_output1));
        CHECK(hipFree(d_output2));
        CHECK(hipFree(d_output3));
        */
        hipDeviceReset();
        return 0;
}

extern "C" int MedianFilt_GPU_main_uint16(unsigned short *Input, unsigned short *Output, int kernel_size, float mu_threshold, int N, int M, int Z)
{
  int deviceCount = -1; // number of devices
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
      fprintf(stderr, "No CUDA devices found\n");
       return -1;
   }
        int ImSize, sizefilter_total, kernel_half_size, midval;
        unsigned short *d_input, *d_output;
        ImSize = N*M*Z;

        CHECK(hipMalloc((void**)&d_input,ImSize*sizeof(unsigned short)));
        CHECK(hipMalloc((void**)&d_output,ImSize*sizeof(unsigned short)));

        CHECK(hipMemcpy(d_input,Input,ImSize*sizeof(unsigned short),hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_output,Input,ImSize*sizeof(unsigned short),hipMemcpyHostToDevice));

	if (Z == 1) {
        /*2D case */
        dim3 dimBlock(BLKXSIZE2D,BLKYSIZE2D);
        dim3 dimGrid(idivup(N,BLKXSIZE2D), idivup(M,BLKYSIZE2D));
        sizefilter_total = (int)(pow(kernel_size,2));
        kernel_half_size = (int)((kernel_size-1)/2);
        midval = (int)(sizefilter_total/2);

        if (kernel_size == 3) medfilt1_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 5) medfilt2_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 7) medfilt3_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else if (kernel_size == 9) medfilt4_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        else medfilt5_kernel_uint16_2D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, ImSize);
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors(hipPeekAtLastError() );
       }
	else {
		    /*3D case*/
        dim3 dimBlock(BLKXSIZE,BLKYSIZE,BLKZSIZE);
        dim3 dimGrid(idivup(N,BLKXSIZE), idivup(M,BLKYSIZE),idivup(Z,BLKXSIZE));
        sizefilter_total = (int)(pow(kernel_size, 3));
        kernel_half_size = (int)((kernel_size-1)/2);
        midval = (int)(sizefilter_total/2);

        if (Z == kernel_size) {
        /* performs operation only on the central frame using all 3D information */
        if (kernel_size == 3) medfilt1_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 5) medfilt2_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 7) medfilt3_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 9) medfilt4_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else medfilt5_pad_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
          }
        else {
        /* Full data (traditional) 3D case */
        if (kernel_size == 3) medfilt1_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 5) medfilt2_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 7) medfilt3_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else if (kernel_size == 9) medfilt4_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
        else medfilt5_kernel_uint16_3D<<<dimGrid,dimBlock>>>(d_input, d_output, kernel_half_size, sizefilter_total, mu_threshold, midval, N, M, Z, ImSize);
            }
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors(hipPeekAtLastError() );
    		}
        CHECK(hipMemcpy(Output,d_output,ImSize*sizeof(unsigned short),hipMemcpyDeviceToHost));
        CHECK(hipFree(d_input));
        CHECK(hipFree(d_output));
        return 0;
}
